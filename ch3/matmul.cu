#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "common.h"

__global__
void MatrixMulKernel(float *M, float *N, float *P, int width)
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    if (row >= width || col >= width)
        return;

    float pval = 0.0f;
    for (int k = 0; k < width; ++ k)
    {
        pval += M[row * width + k] * N[k * width + col];
    }

    P[row * width + col] = pval;
}

int main(int argc, char **argv)
{
    if (argc != 2)
    {
        fprintf(stderr, "Usage: ./matmul <width>\n");
        return;
    }

    int width = atoi(argv[1]);
    int N = width * width;

    dim3 dimGrid(ceil(width / 16.0), ceil(width / 16.0), 1);
    dim3 dimBlock(16, 16, 1);

    // Prepare arguments
    float *M_h = (float *) malloc(N * sizeof(float));
    float *N_h = (float *) malloc(N * sizeof(float));
    float *P_h = (float *) malloc(N * sizeof(float));
    for (int i = 0; i < width; ++ i)
    {
        for (int j = 0; j < width; ++ j)
        {
            M_h[i * width + j] = (i + j) / 2.0;
            N_h[i * width + j] = (j + i) / 2.0;
        }
    }

    // Copy to device
    float *M_d, *N_d, *P_d;
    hipMalloc((void **) &M_d, N * sizeof(float));
    hipMalloc((void **) &N_d, N * sizeof(float));
    hipMalloc((void **) &P_d, N * sizeof(float));

    // Kernel execution
    double callAvg = 0.0;
    double kernelAvg = 0.0;
    int iters = 10;

    for (int i = 0; i < iters; ++ i)
    {
        double start = cpuSecond();
        MatrixMulKernel<<<dimGrid, dimBlock>>>(M_d, N_d, P_d, width);
        double callEnd = cpuSecond();
        hipDeviceSynchronize();
        double kernelEnd = cpuSecond();

        callAvg += callEnd - start;
        kernelAvg += kernelEnd - start;
    }

    callAvg /= iters;
    kernelAvg /= iters;

    // Copy results to host
    hipMemcpy(P_h, P_d, N * sizeof(float), hipMemcpyDeviceToHost);

    // Free memory
    hipFree(M_d);
    hipFree(N_d);
    hipFree(P_d);

    // Print time
    double FLOPS = 2.0 * width * width * width / kernelAvg;

    printf("Kernel launch time:     %10.6lf s\n", callAvg);
    printf("Kernel running time:    %10.6lf s\n", kernelAvg);
    printf("Performance:            %10.6lf GFLOPS\n", FLOPS / 1e9);

    return 0;
}